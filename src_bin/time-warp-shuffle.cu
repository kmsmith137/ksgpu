#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/gputils/Array.hpp"
#include "../include/gputils/CudaStreamPool.hpp"

using namespace std;
using namespace gputils;


// -------------------------------------------------------------------------------------------------


__global__ void shfl_xor_kernel(float *dst, const float *src, int niter)
{
    int s = blockIdx.x * blockDim.x + threadIdx.x;
    float x = src[s];
    
    for (int i = 0; i < niter; i++) {
	x += __shfl_xor_sync(0xffffffff, x, 0x1);
	x += __shfl_xor_sync(0xffffffff, x, 0x2);
	x += __shfl_xor_sync(0xffffffff, x, 0x4);
	x += __shfl_xor_sync(0xffffffff, x, 0x8);
    }

    dst[s] = x;
}


static void time_shfl_xor(int nblocks, int nthreads, int nstreams, int ncallbacks, int niter)
{
    int s = nblocks * nthreads;
    Array<float> dst_arr({nstreams,s}, af_zero | af_gpu);
    Array<float> src_arr({nstreams,s}, af_zero | af_gpu);

    // gigashuffles per callback
    double gsh = 4. * double(s) * double(niter) / pow(2,30.);

    auto callback = [&](const CudaStreamPool &pool, hipStream_t stream, int istream)
	{
	    float *dst = dst_arr.data + istream * s;
	    float *src = src_arr.data + istream * s;
	    
	    shfl_xor_kernel <<<nblocks, nthreads>>> (dst, src, niter);

	    if (pool.num_callbacks == 0)
		return;
	    
	    cout << "    time_shfl_xor [" << pool.num_callbacks
		 << "]: avg time = " << pool.time_per_callback
		 << ", Gshuffles/sec = " << (gsh / pool.time_per_callback)
		 << endl;
	};

    CudaStreamPool pool(callback, ncallbacks, nstreams);
    pool.run();
}


// -------------------------------------------------------------------------------------------------


__global__ void reduce_add_kernel(int *dst, const int *src, int niter)
{
    int s = blockIdx.x * blockDim.x + threadIdx.x;
    int x = src[s];
    
    for (int i = 0; i < niter; i++)
	x = __reduce_add_sync(0xffffffff, x);

    dst[s] = x;
}


static void time_reduce_add(int nblocks, int nthreads, int nstreams, int ncallbacks, int niter)
{
    int s = nblocks * nthreads;
    Array<int> dst_arr({nstreams,s}, af_zero | af_gpu);
    Array<int> src_arr({nstreams,s}, af_zero | af_gpu);

    // gigareduces per callback
    double gre = double(s) * double(niter) / pow(2,30.);

    auto callback = [&](const CudaStreamPool &pool, hipStream_t stream, int istream)
	{
	    int *dst = dst_arr.data + istream * s;
	    int *src = src_arr.data + istream * s;
	    
	    reduce_add_kernel <<<nblocks, nthreads>>> (dst, src, niter);

	    if (pool.num_callbacks == 0)
		return;
	    
	    cout << "    time_reduce_add [" << pool.num_callbacks
		 << "]: avg time = " << pool.time_per_callback
		 << ", Greduces/sec = " << (gre / pool.time_per_callback)
		 << endl;
	};

    CudaStreamPool pool(callback, ncallbacks, nstreams);
    pool.run();
}


// -------------------------------------------------------------------------------------------------


int main(int argc, char **argv)
{
    // (nblocks, nthreads, nstreams, ncallbacks, niter)
    time_shfl_xor(1000, 128, 2, 10, 1000000); 
    time_reduce_add(1000, 128, 2, 10, 3000000);
    return 0;
}
