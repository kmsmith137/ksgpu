#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include "../include/gputils/rand_utils.hpp"
#include "../include/gputils/test_utils.hpp"

using namespace std;

namespace gputils {
#if 0
}  // editor auto-indent
#endif


vector<ssize_t> make_random_strides(int ndim, const ssize_t *shape, int ncontig, int nalign)
{
    assert(ndim <= ArrayMaxDim);
    assert(ncontig >= 0);
    assert(ncontig <= ndim);
    assert(nalign >= 1);

    int nd_strided = ndim - ncontig;
    vector<ssize_t> axis_ordering = rand_permutation(nd_strided);
    
    vector<ssize_t> strides(ndim);
    ssize_t curr_stride = 1;

    for (int d = ndim-1; d >= nd_strided; d--) {
	assert(shape[d] > 0);
	strides[d] = curr_stride;
	curr_stride += (shape[d]-1) * strides[d];
    }

    for (int i = 0; i < nd_strided; i++) {
	int d = axis_ordering[i];
	assert(shape[d] > 0);

	ssize_t smin = (curr_stride + nalign - 1) / nalign;
	ssize_t smax = std::max(smin+1, (2*curr_stride)/nalign);
	strides[d] = nalign * rand_int(smin, smax+1);
	curr_stride += (shape[d]-1) * strides[d];
    }

    return strides;
}


vector<ssize_t> make_random_strides(const vector<ssize_t> &shape, int ncontig, int nalign)
{
    return make_random_strides(shape.size(), &shape[0], ncontig, nalign);
}


// -------------------------------------------------------------------------------------------------

    
void assert_arrays_equal(const Array<float> &arr1,
			 const Array<float> &arr2,
			 const string &name1,
			 const string &name2,
			 const vector<string> &axis_names,
			 float epsabs,
			 float epsrel,
			 ssize_t max_display)
{
    assert(arr1.shape_equals(arr2));
    assert(axis_names.size() == arr1.ndim);
    assert(max_display > 0);
    assert(epsabs >= 0.0);
    assert(epsrel >= 0.0);

    Array<float> harr1 = arr1.to_host(false);  // page_locked=false
    Array<float> harr2 = arr2.to_host(false);  // page_locked=false
    int nfail = 0;

    for (auto ix = arr1.ix_start(); arr1.ix_valid(ix); arr1.ix_next(ix)) {
	float x = harr1.at(ix);
	float y = harr2.at(ix);
	float delta = std::abs(x-y);
	float thresh = epsabs + 0.5*epsrel * (std::abs(x) + std::abs(y));
	    
	if (delta <= thresh)
	    continue;

	if (nfail == 0)
	    cout << "\nassert_arrays_equal() failed [shape=" << arr1.shape_str() << "]\n";

	if (nfail++ >= max_display)
	    continue;
	
	cout << "   ";
	for (int d = 0; d < arr1.ndim; d++)
	    cout << " " << axis_names[d] << "=" << ix[d];

	cout << ": " << name1 << "=" << x << ", " << name2
	     << "=" << y << "  [delta=" << delta << "]\n";
    }
    
    if (nfail > max_display)
	cout << "        [ + " << (nfail-max_display) << " failures]\n";

    if (nfail > 0)
	exit(1);
}


}  // namespace gputils
