#include "hip/hip_runtime.h"
#include <cassert>
#include <complex>
#include <iostream>

#include "../include/gputils/rand_utils.hpp"
#include "../include/gputils/test_utils.hpp"

// is_complex_v<T>, decomplexify_type<T>::type
#include "../include/gputils/complex_type_traits.hpp"

using namespace std;

namespace gputils {
#if 0
}  // editor auto-indent
#endif


// -------------------------------------------------------------------------------------------------


vector<ssize_t> make_random_strides(int ndim, const ssize_t *shape, int ncontig, int nalign)
{
    assert(ndim <= ArrayMaxDim);
    assert(ncontig >= 0);
    assert(ncontig <= ndim);
    assert(nalign >= 1);

    int nd_strided = ndim - ncontig;
    vector<ssize_t> axis_ordering = rand_permutation(nd_strided);
    
    vector<ssize_t> strides(ndim);
    ssize_t curr_stride = 1;

    for (int d = ndim-1; d >= nd_strided; d--) {
	assert(shape[d] > 0);
	strides[d] = curr_stride;
	curr_stride += (shape[d]-1) * strides[d];
    }

    for (int i = 0; i < nd_strided; i++) {
	int d = axis_ordering[i];
	assert(shape[d] > 0);

	ssize_t smin = (curr_stride + nalign - 1) / nalign;
	ssize_t smax = std::max(smin+1, (2*curr_stride)/nalign);
	strides[d] = nalign * rand_int(smin, smax+1);
	curr_stride += (shape[d]-1) * strides[d];
    }

    return strides;
}


vector<ssize_t> make_random_strides(const vector<ssize_t> &shape, int ncontig, int nalign)
{
    return make_random_strides(shape.size(), &shape[0], ncontig, nalign);
}


// -------------------------------------------------------------------------------------------------


template<typename T>
typename gputils::decomplexify_type<T>::type
assert_arrays_equal(const Array<T> &arr1,
		    const Array<T> &arr2,
		    const string &name1,
		    const string &name2,
		    const vector<string> &axis_names,
		    float epsabs,
		    float epsrel,
		    ssize_t max_display,
		    bool verbose)
{
    using Tr = typename decomplexify_type<T>::type;
    
    assert(arr1.shape_equals(arr2));
    assert(axis_names.size() == arr1.ndim);
    assert(max_display > 0);
    assert(epsabs >= 0.0);
    assert(epsrel >= 0.0);

    Array<T> harr1 = arr1.to_host(false);  // page_locked=false
    Array<T> harr2 = arr2.to_host(false);  // page_locked=false
    int nfail = 0;
    Tr maxdiff = 0;

    for (auto ix = arr1.ix_start(); arr1.ix_valid(ix); arr1.ix_next(ix)) {
	T x = harr1.at(ix);
	T y = harr2.at(ix);

	Tr delta;
	if constexpr (!is_unsigned_v<T>)
	    delta = std::abs(x-y);
	else
	    delta = (x > y) ? (x-y) : (y-x);
	
	Tr thresh = 0;
	if constexpr (!is_integral_v<T>)
	    thresh = epsabs + 0.5*epsrel * (std::abs(x) + std::abs(y));

	maxdiff = max(maxdiff, delta);
	bool failed = (delta > thresh);
	
	if (!failed && !verbose)
	    continue;

	if (failed && (nfail == 0))
	    cout << "\nassert_arrays_equal() failed [shape=" << arr1.shape_str() << "]\n";

	if (failed)
	    nfail++;
	
	if ((nfail >= max_display) || !verbose)
	    continue;
	
	cout << "   ";
	for (int d = 0; d < arr1.ndim; d++)
	    cout << " " << axis_names[d] << "=" << ix[d];

	cout << ": " << name1 << "=" << x << ", " << name2
	     << "=" << y << "  [delta=" << delta << "]";

	if (failed)
	    cout << " FAILED";

	cout << "\n";
    }
    
    if ((nfail > max_display) && !verbose)
	cout << "        [ + " << (nfail-max_display) << " more failures]\n";
    
    if (nfail > 0)
	exit(1);
    
    return maxdiff;
}


#define INSTANTIATE_ASSERT_ARRAYS_EQUAL(T)  \
    template				    \
    gputils::decomplexify_type<T>::type	    \
    assert_arrays_equal(		    \
	const Array<T> &arr1,	            \
	const Array<T> &arr2,		    \
	const string &name1,		    \
	const string &name2,		    \
	const vector<string> &axis_names,   \
	float epsabs,                       \
	float epsrel,                       \
	ssize_t max_display, 	            \
	bool verbose);



INSTANTIATE_ASSERT_ARRAYS_EQUAL(float);
INSTANTIATE_ASSERT_ARRAYS_EQUAL(double);
INSTANTIATE_ASSERT_ARRAYS_EQUAL(int);
INSTANTIATE_ASSERT_ARRAYS_EQUAL(long);
INSTANTIATE_ASSERT_ARRAYS_EQUAL(short);
INSTANTIATE_ASSERT_ARRAYS_EQUAL(char);
INSTANTIATE_ASSERT_ARRAYS_EQUAL(unsigned int);
INSTANTIATE_ASSERT_ARRAYS_EQUAL(unsigned long);
INSTANTIATE_ASSERT_ARRAYS_EQUAL(unsigned short);
INSTANTIATE_ASSERT_ARRAYS_EQUAL(unsigned char);
INSTANTIATE_ASSERT_ARRAYS_EQUAL(complex<float>);
INSTANTIATE_ASSERT_ARRAYS_EQUAL(complex<double>);


}  // namespace gputils
