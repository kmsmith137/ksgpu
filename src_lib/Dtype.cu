#include "../include/ksgpu/Dtype.hpp"
#include <sstream>

using namespace std;


namespace ksgpu {
#if 0
}   // pacify editor auto-indent
#endif


// Caller will probably want to wrap output with parentheses.
static string dflag_str(unsigned short f)
{
    const char *sep = "";
    static const char *bar = " | ";
    stringstream ss;
    
    if (!f)
	ss << "0";
    if (f & df_int)
	ss << "df_int"; sep = bar;
    if (f & df_uint)
	ss << sep << "df_uint"; sep = bar;
    if (f & df_float)
	ss << sep << "df_float"; sep = bar;
    if (f & df_complex)
	ss << sep << "df_complex"; sep = bar;

    constexpr unsigned short df_all = df_int | df_uint | df_float | df_complex;
    f &= ~df_all;

    if (f)
	ss << sep << "invalid flags 0x" << hex << f;
    return ss.str();
}


// Pretty-print a dtype.
ostream &operator<<(ostream &os, const Dtype &dt)
{
    if (!dt.is_valid()) {
	os << "invalid_dtype(flags = (" << dflag_str(dt.flags) << "), nbits = " << dt.nbits;
	if (dt.flags == df_complex)
	    os << ", maybe df_complex | df_float was intended?";
	os << ")";
	return os;
    }

    if (dt.flags & df_complex)
	os << ((dt.flags & df_float) ? "complex" : "complex_");
    else if (dt.flags & df_float)
	os << "float";

    if (dt.flags & df_int)
	os << "int";
    else if (dt.flags & df_uint)
	os << "uint";

    if (dt.flags & df_complex)
	os << (dt.nbits/2) << "+" << (dt.nbits/2);
    else
	os << dt.nbits;

    return os;
}


string Dtype::str() const
{
    stringstream ss;
    ss << (*this);
    return ss.str();
}


} // namespace ksgpu
